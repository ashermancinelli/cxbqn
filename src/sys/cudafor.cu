#include "sys_helper.hpp"
#include <cxbqn/cuda/plus.hpp>
#include <hip/hip_runtime.h>
#include <thrust/device_free.h>
#include <thrust/device_malloc.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>

using namespace thrust;

namespace cxbqn::sys {
O<Value> CUDAFor::call(u8 nargs, Args &args) {
  CXBQN_DEBUG("•_CUDAFor: nargs={},args={}", nargs, args);
  auto x = args[1];
  auto f = args[4];
  auto w = args[2];
  auto it = std::find(_runtime->values.begin(), _runtime->values.end(), f);
  auto xind = std::distance(_runtime->values.begin(), it);

  auto xar = dyncast<Array>(x);
  auto war = dyncast<Array>(w);
  if (xar->N() != war->N())
    throw std::runtime_error("•_CUDAFor: 𝕩 and 𝕨 must have matching shapes");
  if (1 == nargs)
    throw std::runtime_error("•_CUDAFor: only dyadic calls");

  const std::size_t N = xar->N();

  thrust::host_vector<f64> hx(N);
  thrust::host_vector<f64> hw(N);
  for (int i = 0; i < N; i++) {
    hx[i] = dyncast<Number>(xar->values[i])->v;
    hw[i] = dyncast<Number>(war->values[i])->v;
  }
  thrust::device_vector<f64> dx = hx;
  thrust::device_vector<f64> dw = hw;
  cuda::Plus<<<1, N>>>(raw_pointer_cast(dx.data()), raw_pointer_cast(dw.data()), N);
  // cuda::Plus<<<1, 5>>>(nullptr, nullptr, 10);
  hx = dx;
  O<Array> ret = CXBQN_NEW(Array, N);
  for (int i = 0; i < N; i++)
    ret->values[i] = CXBQN_NEW(Number, hx[i]);

  return ret;
#if 0
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
  switch (type_builtin(x)) {
  case t_Array: {
    auto ax = dyncast<Array>(x);
    thrust::host_vector<f64> h(ax->N());
    for(int i=0, auto e=ax->values[i]; i<ax->N(); i++, e=ax->values[i]) {
#ifdef CXBQN_DEEPCHECKS
      u8 ee;
      if (t_Number != (ee=type_builtin(e)) and t_Character != ee)
        throw std::runtime_error("•_CUDAFor only accepts non-nested arrays of numbers or chars for 𝕩");
#endif
      h[i] = dyncast<Number>(e)->v;
      thrust::device_vector<f64> d = h;
      // thrust::transform(d.begin(), d.end(), d.begin(), cuda::Plus);
    }
    break;
  }
  case t_Number:
  case t_Character: {
    auto dx = dyncast<Number>(x)->v;
    break;
  }
  default: {
    throw std::runtime_error("•_CUDAFor only accepts data values for 𝕩");
  }
  }
  return CXBQN_NEW(Number, 1);
#endif
}
} // namespace cxbqn::sys
