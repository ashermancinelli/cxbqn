#include "hip/hip_runtime.h"
#include "sys_helper.hpp"
#include <hip/hip_runtime.h>
#include <cxbqn/cuda/all.hpp>
#include <thrust/device_free.h>
#include <thrust/device_malloc.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>

using namespace thrust;

namespace {
void call_inl(std::size_t rt_idx, f64 *x, f64 *w, std::size_t N) {
  // +-×÷⋆√⌊⌈|¬
  switch (rt_idx) {
  case 0: cxbqn::cuda::plus<<<1, N>>>(x, w, N); return;
  case 1: cxbqn::cuda::minus<<<1, N>>>(x, w, N); return;
  case 2: cxbqn::cuda::mul<<<1, N>>>(x, w, N); return;
  case 3: cxbqn::cuda::div<<<1, N>>>(x, w, N); return;
  case 4: cxbqn::cuda::power<<<1, N>>>(x, w, N); return;
  case 5: cxbqn::cuda::root<<<1, N>>>(x, w, N); return;
  case 6: cxbqn::cuda::floor<<<1, N>>>(x, w, N); return;
  case 7: cxbqn::cuda::ceil<<<1, N>>>(x, w, N); return;
  case 8: cxbqn::cuda::stile<<<1, N>>>(x, w, N); return;
  case 9: cxbqn::cuda::not_<<<1, N>>>(x, w, N); return;

  // ∧∨<>≠=≤≥≡≢
  // case 9: cxbqn::cuda::not_<<<1, N>>>(x, w, N); return;

  default:
    throw std::runtime_error("•_CUDAFor: 𝕗 unsupported in device code");
  }
}
} // namespace

namespace cxbqn::sys {
O<Value> CUDAFor::call(u8 nargs, Args &args) {
  CXBQN_DEBUG("•_CUDAFor: nargs={},args={}", nargs, args);
  if (1 == nargs)
    throw std::runtime_error("•_CUDAFor: only dyadic calls");
  auto x = args[1];
  auto f = args[4];
  auto w = args[2];
  auto it = std::find(_runtime->values.begin(), _runtime->values.end(), f);
  auto xind = std::distance(_runtime->values.begin(), it);

  thrust::device_vector<f64> hx;
  thrust::device_vector<f64> hw;

  int N=-1;
  bool isxar=false,iswar=false;
  std::vector<uz> sh;

  if (isxar = (t_Array == type_builtin(x))) {
    auto xar = dyncast<Array>(x);
    N = xar->N();
    sh = xar->shape;
    hx.resize(N);
    hw.resize(N);
    for (int i = 0; i < N; i++)
      hx[i] = dyncast<Number>(xar->values[i])->v;
    N = xar->N();
  } 
  if (iswar = (t_Array == type_builtin(w))) {
    auto war = dyncast<Array>(w);
    if (N<0) {
      N = war->N();
      sh = war->shape;
      hx.resize(N);
      hw.resize(N);
    }
    else if (N != war->N())
      throw std::runtime_error("•_CUDAFor: array lengths must be the same");
    for (int i = 0; i < N; i++)
      hw[i] = dyncast<Number>(war->values[i])->v;
  } 

  if (N < 0) {
    N = 1;
    hx.resize(N);
    hw.resize(N);
    sh.push_back(1);
  }

  if (not isxar) {
    auto v = dyncast<Number>(x)->v;
    for (int i = 0; i < N; i++)
      hx[i] = v;
  }
  if (not iswar) {
    auto v = dyncast<Number>(w)->v;
    for (int i = 0; i < N; i++)
      hw[i] = v;
  }

  thrust::device_vector<f64> dx = hx;
  thrust::device_vector<f64> dw = hw;
  call_inl(xind, raw_pointer_cast(dx.data()), raw_pointer_cast(dw.data()), N);
  hx = dx;

  O<Array> ret = CXBQN_NEW(Array, N);
  ret->shape = sh;
  for (int i = 0; i < N; i++)
    ret->values[i] = CXBQN_NEW(Number, hx[i]);

  return ret;
}
} // namespace cxbqn::sys
